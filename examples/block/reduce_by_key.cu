#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 Savely Pototsky (SavaLione)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 * 3. Neither the name of the University nor the names of its contributors
 *    may be used to endorse or promote products derived from this software
 *    without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE REGENTS AND CONTRIBUTORS ``AS IS'' AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE REGENTS OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS
 * OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
 * HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
 * OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGE.
 */

#include <newcub/cub.cuh>


template <
    int         BLOCK_THREADS,          ///< Number of CTA threads
    typename    KeyT,                   ///< Key type
    typename    ValueT>                 ///< Value type
__global__ void Kernel()
{
    // Tuple type for scanning (pairs accumulated segment-value with segment-index)
    typedef hipcub::KeyValuePair<int, ValueT> OffsetValuePairT;

    // Reduce-value-by-segment scan operator
    typedef hipcub::ReduceBySegmentOp<hipcub::Sum> ReduceBySegmentOpT;

    // Parameterized BlockDiscontinuity type for setting head flags
    typedef hipcub::BlockDiscontinuity<
            KeyT,
            BLOCK_THREADS>
        BlockDiscontinuityKeysT;

    // Parameterized BlockScan type
    typedef hipcub::BlockScan<
            OffsetValuePairT,
            BLOCK_THREADS,
            hipcub::BLOCK_SCAN_WARP_SCANS>
        BlockScanT;

    // Shared memory
    __shared__ union TempStorage
    {
        typename BlockScanT::TempStorage                scan;           // Scan storage
        typename BlockDiscontinuityKeysT::TempStorage   discontinuity;  // Discontinuity storage
    } temp_storage;


    // Read data (each thread gets 3 items each, every 9 items is a segment)
    KeyT    my_keys[3]      = {threadIdx.x / 3, threadIdx.x / 3, threadIdx.x / 3};
    ValueT  my_values[3]    = {1, 1, 1};

    // Set head segment head flags
    int     my_flags[3];
    BlockDiscontinuityKeysT(temp_storage.discontinuity).FlagHeads(
        my_flags,
        my_keys,
        hipcub::Inequality());

    __syncthreads();






}
